#include <hip/hip_runtime.h>
#include <iostream>
#include "cpu_rasterizer.h"

int main(int argc, char const *const *argv) {
    // CPU
    hipEvent_t start, stop;
    float milliseconds = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    cpu_render();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // TODO GPU
    return 0;
}